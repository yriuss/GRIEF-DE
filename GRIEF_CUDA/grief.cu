#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009-2010, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/
#include "grief.h"
#include "precomp.hpp"
#include <algorithm>
#include <vector>
#include <iostream>
#include <iomanip>

namespace cv
{
namespace xfeatures2d
{





float evaluation(Eigen::MatrixXd individual){
	return -1;
}


Ptr<GriefDescriptorExtractor> GriefDescriptorExtractor::create(int bytes, bool use_orientation, EvalFunction evaluation, int N_pop, 
															   float cr, float jr, float F, int mutation_algorithm, int crossover_algorithm)
{
	return makePtr<GriefDescriptorExtractorImpl>(bytes, use_orientation, evaluation, N_pop, cr, jr, F, mutation_algorithm, crossover_algorithm);
}
#include <unistd.h>
int GriefDescriptorExtractorImpl::load(std::string fileName) {
	std::string CURRENT_DIR = get_current_dir_name();
	using namespace std;
	ifstream file(CURRENT_DIR +"/../GRIEF_CUDA/" + fileName);

	std::string line;
	uint16_t i = 0, j = 0;
	bool successful=false;
	std::string cell;
	//std::cout << CURRENT_DIR +"../GRIEF_CUDA/" + fileName;
	while (std::getline(file, line)) {
		//std::cout << line;
		std::vector<int> v;
		istringstream is(line);
		while (std::getline(is, cell, ' ')) {
			//std::cout << std::stoi(cell);
			individual[i][j] = std::stoi(cell);
			j++;
		}
		successful=true;
		i++;
		j = 0;
	}
	
	file.close();
	return successful;
}

float GriefDescriptorExtractorImpl::get_b_fit(){
	return get_best_fit();
}

float GriefDescriptorExtractor::get_b_fit(){
	return 1;
}

__device__ int smoothedSum(cuda::PtrStepSz<int> sum, int x, int y, int _x, int _y)
{
	static const int HALF_KERNEL = GriefDescriptorExtractorImpl::KERNEL_SIZE / 2;
	//printf("ajsdihji%d\n", sum(1,1));
	//printf("passou aqui%d\n", x);
	int img_y = (int)(y + 0.5) + _y;
	int img_x = (int)(x + 0.5) + _x;
	return   sum(img_y + HALF_KERNEL + 1, img_x + HALF_KERNEL + 1)
		   - sum(img_y + HALF_KERNEL + 1, img_x - HALF_KERNEL)
		   - sum(img_y - HALF_KERNEL, img_x + HALF_KERNEL + 1)
		   + sum(img_y - HALF_KERNEL, img_x - HALF_KERNEL);
	//printf("%d e %d\n", individual[0][0], individual[0][0]);
	//printf("passou aqui%d\n", individual[0][0]);
}

__global__ void compare_results(uchar* desc, arr2 * result){
	//printf("%d\n", result[0][0]);
	desc[blockIdx.x] += (result[threadIdx.x][0] <  result[threadIdx.x][1])<< (7 - threadIdx.x);
	//printf("%d\n", desc[blockIdx.x]);
}

static void pixelTests16(InputArray _sum, const std::vector<KeyPoint>& keypoints, OutputArray _descriptors, bool use_orientation, int individual[512][4])
{
	Matx21f R;
	Mat sum = _sum.getMat(), descriptors = _descriptors.getMat();
	int result;
	for (size_t i = 0; i < keypoints.size(); ++i)
	{
		uchar* desc = descriptors.ptr(static_cast<int>(i));
		const KeyPoint& pt = keypoints[i];
		if ( use_orientation )
		{
		  float angle = pt.angle;
		  angle *= (float)(CV_PI/180.f);
		  R(0,0) = sin(angle);
		  R(1,0) = cos(angle);
		}

//#include "generated_16.i"
	}
}

static void pixelTests32(InputArray _sum, const std::vector<KeyPoint>& keypoints, OutputArray _descriptors, bool use_orientation, int individual[512][4])
{
	Matx21f R;
	
	Mat sum = _sum.getMat(), descriptors = _descriptors.getMat();
	exit(-1);
	int result;
	for (size_t i = 0; i < keypoints.size(); ++i)
	{
		uchar* desc = descriptors.ptr(static_cast<int>(i));
		const KeyPoint& pt = keypoints[i];
		if ( use_orientation )
		{
		  float angle = pt.angle;
		  angle *= (float)(CV_PI / 180.f);
		  R(0,0) = sin(angle);
		  R(1,0) = cos(angle);
		}

//#include "generated_32.i"
	}
}

__global__ static void pixelTests64_kernel(cuda::PtrStepSz<int> sum, float* x, float* y,cuda::PtrStepSz<uchar>  descriptors, bool* use_orientation, arr4* individual){
	//Matx21f R;
	//arr2* result_child = nullptr;
	//arr4* individual_child = nullptr;
	//hipMalloc(&result_child, (sizeof(int) *512*2));
	//hipMalloc(&result_child2, (sizeof(int) *512*2));
	//hipMemcpyAsync(result_child, result, (sizeof(int)*512*2), hipMemcpyDeviceToDevice);

	//hipMalloc(&individual_child, (sizeof(int) *512*4));
	//hipMemcpyAsync(individual_child, individual, (sizeof(int)*512*4), hipMemcpyDeviceToDevice);
	
	
	//hipMalloc(&desc_child, (sizeof(uchar) *64));
	//printf("%d\n", individual[0][0]);

	//KeyPoint& pt = ;
	//if ( use_orientation )
	//{
	//  //float angle = pt.angle;
	//  //angle *= (float)(CV_PI/180.f);
	//  //R(0,0) = sin(angle);
	//  //R(1,0) = cos(angle);
	//}
	//printf("passou aqui%d\n", sum(1,1));
	//desc[0] = 5;
	//smoothedSum<<<512,2>>>(sum, x[blockIdx.x], y[blockIdx.x], individual_child, result_child);
	//hipDeviceSynchronize();
	//memcpy(result, result_child, (sizeof(int)*512*2));
	//memcpy(result_child2, result, (sizeof(int)*512*2));
	//printf("%d\n", result[0][0]);
	//printf("opa%d", result[0][0]);
	//compare_results<<<64,8>>>(desc_child, result_child);
	//memcpy(desc, desc_child, (sizeof(int)*512*2));
	//hipDeviceSynchronize();
	//desc[0] = 1;
	//printf("%d\n", desc[0]);
	descriptors(blockIdx.x,threadIdx.x) = ((smoothedSum(sum, x[blockIdx.x], y[blockIdx.x], individual[8*threadIdx.x][0], individual[8*threadIdx.x][1]) < smoothedSum(sum, x[blockIdx.x], y[blockIdx.x], individual[8*threadIdx.x][2], individual[8*threadIdx.x][3])) << 7)
	+((smoothedSum(sum, x[blockIdx.x], y[blockIdx.x], individual[8*threadIdx.x + 1][0], individual[8*threadIdx.x + 1][1]) < smoothedSum(sum, x[blockIdx.x], y[blockIdx.x], individual[8*threadIdx.x + 1][2], individual[8*threadIdx.x + 1][3])) << 6)
	+((smoothedSum(sum, x[blockIdx.x], y[blockIdx.x], individual[8*threadIdx.x + 2][0], individual[8*threadIdx.x + 2][1]) < smoothedSum(sum, x[blockIdx.x], y[blockIdx.x], individual[8*threadIdx.x + 2][2], individual[8*threadIdx.x + 2][3])) << 5)
	+((smoothedSum(sum, x[blockIdx.x], y[blockIdx.x], individual[8*threadIdx.x + 3][0], individual[8*threadIdx.x + 3][1]) < smoothedSum(sum, x[blockIdx.x], y[blockIdx.x], individual[8*threadIdx.x + 3][2], individual[8*threadIdx.x + 3][3])) << 4)
	+((smoothedSum(sum, x[blockIdx.x], y[blockIdx.x], individual[8*threadIdx.x + 4][0], individual[8*threadIdx.x + 4][1]) < smoothedSum(sum, x[blockIdx.x], y[blockIdx.x], individual[8*threadIdx.x + 4][2], individual[8*threadIdx.x + 4][3])) << 3)
	+((smoothedSum(sum, x[blockIdx.x], y[blockIdx.x], individual[8*threadIdx.x + 5][0], individual[8*threadIdx.x + 5][1]) < smoothedSum(sum, x[blockIdx.x], y[blockIdx.x], individual[8*threadIdx.x + 5][2], individual[8*threadIdx.x + 5][3])) << 2)
	+((smoothedSum(sum, x[blockIdx.x], y[blockIdx.x], individual[8*threadIdx.x + 6][0], individual[8*threadIdx.x + 6][1]) < smoothedSum(sum, x[blockIdx.x], y[blockIdx.x], individual[8*threadIdx.x + 6][2], individual[8*threadIdx.x + 6][3])) << 1)
	+((smoothedSum(sum, x[blockIdx.x], y[blockIdx.x], individual[8*threadIdx.x + 7][0], individual[8*threadIdx.x + 7][1]) < smoothedSum(sum, x[blockIdx.x], y[blockIdx.x], individual[8*threadIdx.x + 7][2], individual[8*threadIdx.x + 7][3])));// << 0
//#include "generated_64.i"
}

Eigen::MatrixXd GriefDescriptorExtractorImpl::get_best_indv(){
	return get_best_ind();
}

Eigen::MatrixXd GriefDescriptorExtractor::get_best_indv(){
	return Eigen::MatrixXd(1,1);
}

void GriefDescriptorExtractorImpl::pixelTests64(InputArray sum, const std::vector<KeyPoint>& keypoints,cuda::GpuMat&  descriptors, bool use_orientation)
{
	//auto start = std::chrono::high_resolution_clock::now();
	cuda::GpuMat _sum;
	_sum.upload(sum.getMat());
	bool* _use_orientation;
	
	std::vector<KeyPoint> keypoints_aux  = keypoints;
	KeyPoint *arr_keypoints = &keypoints_aux[0], *gpu_keypoints;

	float*x, *y;
	float*_x, *_y;

	x = (float*)malloc(sizeof(float)*keypoints.size());
	y = (float*)malloc(sizeof(float)*keypoints.size());
	
	for(int i = 0; i < keypoints.size(); i++){
		x[i] = keypoints[i].pt.x;
		y[i] = keypoints[i].pt.y;
	}
	
	//printf("passou aqui%f\n", x[2]);
	//arr_keypoints = (KeyPoint*)*malloc(sizeof(KeyPoint));
	arr4* gpu_individual;
	
	//cuda::GpuMat _descriptors(keypoints.size(), 64, CV_8UC1);
	
	//hipMalloc((void **)&_descriptors, sizeof(cv::KeyPoint));
	hipMalloc((void **)&gpu_individual,sizeof(int)*512*4);
	
	hipMalloc((void **)&_x,sizeof(float)*keypoints.size());
	hipMalloc((void **)&_y,sizeof(float)*keypoints.size());
	//hipMallocManaged(&_sum,sizeof(Mat)*keypoints.size());

	hipMemcpy(_x, x, sizeof(float)*keypoints.size(), hipMemcpyHostToDevice);
	hipMemcpy(_y, y, sizeof(float)*keypoints.size(), hipMemcpyHostToDevice);
	

	//hipMemcpy(_use_orientation, &use_orientation, sizeof(bool), hipMemcpyHostToDevice);
	
	hipMemcpy(gpu_individual, individual, sizeof(int)*512*4, hipMemcpyHostToDevice);
	//hipMemcpy(_descriptors, &aux, sizeof(cuda::GpuMat), hipMemcpyHostToDevice);
	
	
	
	
	//hipMemcpy(_sum, &aux, sizeof(KeyPoint)*keypoints.size(), hipMemcpyHostToDevice);
	pixelTests64_kernel<<<keypoints.size(),64>>>(_sum, _x, _y,descriptors, _use_orientation, gpu_individual);
	hipDeviceSynchronize();
	
	hipFree(gpu_individual); hipFree(_x); hipFree(_y);
	//std::cout << descriptors.size() << std::endl;
	//hipMemcpy(aux, _descriptors, sizeof(Mat), hipMemcpyDeviceToHost);

	//Mat a;
	//_descriptors.download(a);
	//descriptors.assign(a);
	//std::cout << a << std::endl;
	//std::cout << descriptors.getMat().size() << std::endl;
	//auto finish = std::chrono::high_resolution_clock::now();
	//std::chrono::duration<double, std::micro> elapsed = finish - start;
	//std::cout << "elapsed " << elapsed.count() << std::endl;
	
	//exit(-1);
	
}



void GriefDescriptorExtractorImpl::evolve(uint ng){
	
	for(int g = 0; g < ng; g++){
		
		auto start = std::chrono::high_resolution_clock::now();
		for(int i = 0; i < N_pop; i++){
			
			mutate(i);
			
			crossover(i);
			
			if(is_infeasible())
				repair(i);
			
			selection(i);
			//std::cout << i << std::endl;

		}//exit(-1);
		
		std::cout <<  get_best_fit() << std::endl;
		
		bfit.emplace_back(get_best_fit());
		auto finish = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double, std::milli> elapsed = finish - start;
		std::cout << "Gen " << g+1 << ": Elapsed time: " << elapsed.count() << " ms." << std::endl;
		
		
	}
}

void GriefDescriptorExtractor::evolve(uint ng){
	
}

std::vector<float> GriefDescriptorExtractorImpl::gbfit(){
	return bfit;
}

std::vector<float> GriefDescriptorExtractor::gbfit(){
	return std::vector<float>{0,0};
}



//void GriefDescriptorExtractor::plot_convergence(){
//}

GriefDescriptorExtractorImpl::GriefDescriptorExtractorImpl( int bytes, bool use_orientation, EvalFunction evaluation, 
															int N_pop, float cr, float jr, float F, int mutation_algorithm, int crossover_algorithm) :
	bytes_(bytes), 
	DE(N_pop, std::vector<int>{bytes*8, 4}, cr, jr, evaluation, F, MAXIMIZATION, std::vector<int>{-24, 24}, mutation_algorithm, crossover_algorithm)
{
	this->N_pop = N_pop;
	this->jr = jr;

	load("test_pairs.brief");
	
	use_orientation_ = use_orientation;
	switch (bytes)
	{
		case 16:
			//test_fn_ = pixelTests16;
			break;
		case 32:
			//test_fn_ = pixelTests32;
			break;
		case 64:
			//test_fn_ = pixelTests64;
			break;
		default:
			CV_Error(Error::StsBadArg, "bytes must be 16, 32, or 64");
	}
	}

int GriefDescriptorExtractorImpl::descriptorSize() const
{
	return bytes_;
}

int GriefDescriptorExtractorImpl::descriptorType() const
{
	return CV_8UC1;
}

int GriefDescriptorExtractorImpl::defaultNorm() const
{
	return NORM_HAMMING;
}

void GriefDescriptorExtractorImpl::read( const FileNode& fn)
{
	int dSize = fn["descriptorSize"];
	switch (dSize)
	{
		case 16:
			//test_fn_ = pixelTests16;
			break;
		case 32:
			//test_fn_ = pixelTests32;
			break;
		case 64:
			//test_fn_ = pixelTests64;
			break;
		default:
			CV_Error(Error::StsBadArg, "descriptorSize must be 16, 32, or 64");
	}
	bytes_ = dSize;
}

void GriefDescriptorExtractorImpl::write( FileStorage& fs) const
{
	fs << "descriptorSize" << bytes_;
}

void GriefDescriptorExtractorImpl::getInd(){
	for(int i = 0; i < 512; i++){
		for(int j=0; j < 4; j++)
			std::cout << individual[i][j] << " ";
		std::cout << std::endl;
	}

}

void GriefDescriptorExtractor::getInd(){
	
}

void GriefDescriptorExtractorImpl::compute(InputArray image,
										   std::vector<KeyPoint>& keypoints,
										   cuda::GpuMat& descriptors)
{
	// Construct integral image for fast smoothing (box filter)
	Mat sum;

	Mat grayImage = image.getMat();
	if( image.type() != CV_8U ) cvtColor( image, grayImage, COLOR_BGR2GRAY );

	///TODO allow the user to pass in a precomputed integral image
	//if(image.type() == CV_32S)
	//  sum = image;
	//else

	integral( grayImage, sum, CV_32S);

	//Remove keypoints very close to the border
	KeyPointsFilter::runByImageBorder(keypoints, image.size(), PATCH_SIZE/2 + KERNEL_SIZE/2);
	
	descriptors.create((int)keypoints.size(), bytes_, CV_8U);
	descriptors.setTo(Scalar::all(0));
	pixelTests64(sum, keypoints, descriptors, use_orientation_);
}

void GriefDescriptorExtractor::compute(InputArray image,
										   std::vector<KeyPoint>& keypoints,
										   cuda::GpuMat& descriptors)
{
}

void GriefDescriptorExtractorImpl::setInd(Eigen::MatrixXd new_individual){
	load("test_pairs.brief");
	std::cout << individual[0][0] << individual[0][1] << individual[0][2] << individual[0][3]  << std::endl;
	//for(int i = 0; i < bytes_*8; i++){
	//	for(int j=0; j<4; j++){
	//		individual[i][j] = new_individual(i,j);
	//	}
	//}
}

void GriefDescriptorExtractor::setInd(Eigen::MatrixXd new_individual){
}



}
} // namespace cv